#include "hip/hip_runtime.h"

#include "kernel.h"
#include <hip/hip_runtime.h>
#include <list>
#include <string>
#include <assert.h>
#include <omp.h>

using namespace::std;


typedef struct KernelVectorAddCBTrigILP2Params
{
public:
	dim3 m_bs;
	dim3 m_gs;
	int m_NumberOfElements;

	KernelVectorAddCBTrigILP2Params(int bsx, int bsy, int bsz, int gsx, int gsy, int gsz, int numele) :
		m_bs(bsx, bsy, bsz),
		m_gs(gsx, gsy, gsz),
		m_NumberOfElements(numele)
	{
		if (bsx < 1) { printf("\n***Error bsx < 1\n"); exit(EXIT_FAILURE); }
		if (bsx > 1024) { printf("\n***Error bsx > 1024\n"); exit(EXIT_FAILURE); }
		if (bsy != 1) { printf("\n***Error bsy != 1\n"); exit(EXIT_FAILURE); }
		if (bsz != 1) { printf("\n***Error bsz != 1\n"); exit(EXIT_FAILURE); }

		if (gsx < 1) { printf("\n***Error gsx < 1\n"); exit(EXIT_FAILURE); }
		if (gsy != 1) { printf("\n***Error gsy != 1\n"); exit(EXIT_FAILURE); }
		if (gsz != 1) { printf("\n***Error gsz != 1\n"); exit(EXIT_FAILURE); }
		if (numele < 1) { printf("\n***Error numele < 1\n"); exit(EXIT_FAILURE); }
	}

} KernelVectorAddCBTrigILP2Params_t;

void QueryKernelVectorAddCBTrigILP2(char *KernelName, int gs_start, int gs_end, int gs_inc, int numele)
{
	list<KernelVectorAddCBTrigILP2Params_t*> params;

	for (int gsx = gs_start; gsx < gs_end; gsx += gs_inc)
		for (int bsx = 32; bsx < 1025; bsx += 32)
			params.push_back(new KernelVectorAddCBTrigILP2Params_t(bsx, 1, 1, gsx, 1, 1, numele));

	printf("#\n# %s\n#", KernelName);
	list<KernelVectorAddCBTrigILP2Params_t*>::iterator i = params.begin();
	printf("\n%s:   compile: params -bs %4d,%d,%d -gs %4d,%d,%d -numele %d",
		KernelName,
		(*i)->m_bs.x,
		(*i)->m_bs.y,
		(*i)->m_bs.z,
		(*i)->m_gs.x,
		(*i)->m_gs.y,
		(*i)->m_gs.z,
		(*i)->m_NumberOfElements);

	for (i++; i != params.end(); ++i)
	{
		printf("\n%s: nocompile: params -bs %4d,%d,%d -gs %4d,%d,%d -numele %d",
			KernelName,
			(*i)->m_bs.x,
			(*i)->m_bs.y,
			(*i)->m_bs.z,
			(*i)->m_gs.x,
			(*i)->m_gs.y,
			(*i)->m_gs.z,
			(*i)->m_NumberOfElements);
	}
	printf("\n");
}

//
// compute bound version of vector add kernel
//
__global__ void
kernelVectorAddCBTrigILP2(const float *A, const float *B, float *C, float K1, float K2, int numElements)
{
	const int s1 = blockDim.x * gridDim.x;
	const int stride = blockDim.x * gridDim.x * 2;

	for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < numElements; i += stride)
	{
		// C[i] = sin(K1)*A[i] + cos(K2)*B[i]
		// C[i+s] = sin(K1)*A[i+s] + cos(K2)*B[i+s]
		float T1 = A[i];
		float T2 = B[i];
		float T3 = i + s1 < numElements ? A[i + s1] : 0;
		float T4 = i + s1 < numElements ? B[i + s1] : 0;

		float T5 = sin(K1);
		float T6 = cos(K2);
		float T7 = sin(K1);
		float T8 = cos(K2);

		float T9  = T1*T5;
		float T10 = T2*T6;
		float T11 = T3*T7;
		float T12 = T4*T8;

		C[i] = T9 + T10;
		if (i + s1 < numElements) C[i + s1] = T11 + T12;
	}
}

void LaunchKernelVectorAddCBTrigILP2(dim3& gs, dim3& bs, char **argv, int argc, int nextarg)
{
	printf("\nPreparing %s", KernelVectorAddCBTrigILP2Name);
	if (strcmp(argv[nextarg], "-numele") == 0)
	{
		printf("\nAllocating RAM");

		hipError_t err = hipSuccess;
		int numElements = stoi(argv[nextarg + 1], nullptr);
		size_t size = numElements * sizeof(float);

		KernelVectorAddCBTrigILP2Params_t Verify(bs.x, bs.y, bs.z, gs.x, gs.y, gs.z, numElements);

		float *h_B = new float[numElements];
		float *h_C = new float[numElements];

		// Verify that allocations succeeded
		if (h_B == NULL || h_C == NULL)
		{
			printf("Failed to allocate host vectors in LaunchKernelVectorAddCBTrigILP2\n");
			exit(EXIT_FAILURE);
		}


		float *d_A = NULL;
		err = hipMalloc((void **)&d_A, size);
		if (err != hipSuccess)
		{
			printf("Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}

		float *d_B = NULL;
		err = hipMalloc((void **)&d_B, size);
		if (err != hipSuccess)
		{
			printf("Failed to allocate device vector B (error code %s)!\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}

		float *d_C = NULL;
		err = hipMalloc((void **)&d_C, size);
		if (err != hipSuccess)
		{
			printf("Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}

		printf("\nInitializing GPU RAM");
		InitRandomSequence(d_A, numElements);
		InitRandomSequence(d_B, numElements);

		printf("\nLaunching kernel: kernelVectorAddCBTrigILP2");
		printf("\n\tgridsize  (%d,%d,%d)", gs.x, gs.y, gs.z);
		printf("\n\tblocksize (%d,%d,%d)", bs.x, bs.y, bs.z);
		printf("\n\tNumElements %d", numElements);

		kernelVectorAddCBTrigILP2 << <gs, bs >> > (d_A, d_B, d_C, 0.0f, 0.0f, numElements);
		hipDeviceSynchronize();

		err = hipGetLastError();
		if (err != hipSuccess)
		{
			printf("Failed to launch kernelVectorAddCBTrigILP2 (error code %s)!\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}

		err = hipMemcpy(h_B, d_B, size, hipMemcpyDeviceToHost);
		if (err != hipSuccess)
		{
			printf("Failed to copy vector B from device to host (error code %s)!\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}
		err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
		if (err != hipSuccess)
		{
			printf("Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}

		// Verify that the result vector is correct
		printf("\nValidating results ...");
#pragma omp parallel for
		for (int i = 0; i < numElements; ++i)
		{
			if (fabs(h_B[i] - h_C[i]) > 1e-5)
			{
				printf("Result verification failed at element %d!\n", i);
				exit(EXIT_FAILURE);
			}
		}
		printf(" success!\n");

		err = hipFree(d_A);
		if (err != hipSuccess)
		{
			printf("Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}

		err = hipFree(d_B);
		if (err != hipSuccess)
		{
			printf("Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}

		err = hipFree(d_C);
		if (err != hipSuccess)
		{
			printf("Failed to free device vector C (error code %s)!\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}

		delete[]h_B;
		delete[]h_C;
	}
	else
	{
		printf("\nExpecting -numele, but saw %s", argv[nextarg]);
		exit(EXIT_FAILURE);
	}
}