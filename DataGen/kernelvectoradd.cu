#include "hip/hip_runtime.h"

#include "kernel.h"
#include <hip/hip_runtime.h>
#include <list>
#include <string>
#include <assert.h>
#include <omp.h>

using namespace::std;


typedef struct KernelVectorAddParams
{
public:
	dim3 m_bs;
	dim3 m_gs;
	int m_NumberOfElements;

	KernelVectorAddParams(int bsx, int bsy, int bsz, int gsx, int gsy, int gsz, int numele) :
		m_bs(bsx,bsy,bsz), 
		m_gs(gsx,gsy,gsz),
		m_NumberOfElements(numele)
	{
		if (bsx < 1)	{ printf("\n***Error bsx < 1\n"); exit(EXIT_FAILURE); }
		if (bsx > 1024) { printf("\n***Error bsx > 1024\n"); exit(EXIT_FAILURE); }
		if (bsy != 1)	{ printf("\n***Error bsy != 1\n"); exit(EXIT_FAILURE); }
		if (bsz != 1)	{ printf("\n***Error bsz != 1\n"); exit(EXIT_FAILURE); }

		if (gsx < 1)	{ printf("\n***Error gsx < 1\n"); exit(EXIT_FAILURE); }
		if (gsy != 1)	{ printf("\n***Error gsy != 1\n"); exit(EXIT_FAILURE); }
		if (gsz != 1)	{ printf("\n***Error gsz != 1\n"); exit(EXIT_FAILURE); }
		if (numele < 1) { printf("\n***Error numele < 1\n"); exit(EXIT_FAILURE); }
	}

} KernelVectorAddParams_t;

void QueryKernelVectorAdd(char *KernelName, int gs_start, int gs_end, int gs_inc, int numele)
{

	list<KernelVectorAddParams_t*> params;

	for (int gsx = gs_start; gsx < gs_end; gsx += gs_inc)
		for (int bsx = 32; bsx < 1025; bsx += 32)
			params.push_back(new KernelVectorAddParams_t(bsx, 1, 1, gsx, 1, 1, numele));

	printf("\n#\n# %s\n#", KernelName);
	list<KernelVectorAddParams_t*>::iterator i = params.begin();
	printf("\n%s:   compile: params -bs %4d,%d,%d -gs %4d,%d,%d -numele %d",
		KernelName,
		(*i)->m_bs.x,
		(*i)->m_bs.y,
		(*i)->m_bs.z,
		(*i)->m_gs.x,
		(*i)->m_gs.y,
		(*i)->m_gs.z,
		(*i)->m_NumberOfElements);

	for (i++; i != params.end(); ++i) 
	{
		printf("\n%s: nocompile: params -bs %4d,%d,%d -gs %4d,%d,%d -numele %d",
			KernelName,
			(*i)->m_bs.x,
			(*i)->m_bs.y,
			(*i)->m_bs.z,
			(*i)->m_gs.x,
			(*i)->m_gs.y,
			(*i)->m_gs.z,
			(*i)->m_NumberOfElements);
	}
	printf("\n");
}

__global__ void
kernelVectorAdd(const float *A, const float *B, float *C, int numElements)
{
	const int stride = blockDim.x * gridDim.x;
	const int tid = blockDim.x * blockIdx.x + threadIdx.x;

	for (int i = tid; i < numElements; i += stride)
	{
		C[i] = A[i] + B[i];
	}
}

void LaunchKernelVectorAdd(dim3& gs, dim3& bs, char **argv, int argc, int nextarg)
{
	printf("\nPreparing %s", KernelVectorAddName);
	if (strcmp(argv[nextarg], "-numele") == 0)
	{
		printf("\nAllocating RAM");

		hipError_t err = hipSuccess;
		int numElements = stoi(argv[nextarg + 1], nullptr);
		size_t size = numElements * sizeof(float);

		KernelVectorAddParams_t Verify(bs.x, bs.y, bs.z, gs.x, gs.y, gs.z, numElements);

		float *h_A = new float[numElements];
		float *h_B = new float[numElements];
		float *h_C = new float[numElements];

		// Verify that allocations succeeded
		if (h_A == NULL || h_B == NULL || h_C == NULL)
		{
			printf("Failed to allocate host vectors in LaunchVectorAddKernel\n");
			exit(EXIT_FAILURE);
		}

		float *d_A = NULL;
		err = hipMalloc((void **)&d_A, size);
		if (err != hipSuccess)
		{
			printf("Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}

		float *d_B = NULL;
		err = hipMalloc((void **)&d_B, size);
		if (err != hipSuccess)
		{
			printf("Failed to allocate device vector B (error code %s)!\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}

		float *d_C = NULL;
		err = hipMalloc((void **)&d_C, size);
		if (err != hipSuccess)
		{
			printf("Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}

		printf("\nInitializing GPU RAM");
		InitRandomSequence(d_A, numElements);
		InitRandomSequence(d_B, numElements);

		//printf("\nCopying HOST RAM to GPU");
		//err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
		//if (err != hipSuccess)
		//{
		//	printf("Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
		//	exit(EXIT_FAILURE);
		//}

		//err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
		//if (err != hipSuccess)
		//{
		//	printf("Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
		//	exit(EXIT_FAILURE);
		//}

		printf("\nLaunching kernel: kernelVectorAdd");
		printf("\n\tgridsize  (%d,%d,%d)", gs.x, gs.y, gs.z);
		printf("\n\tblocksize (%d,%d,%d)", bs.x, bs.y, bs.z);
		printf("\n\tNumElements %d", numElements);

		kernelVectorAdd << <gs, bs >> > (d_A, d_B, d_C, numElements);
		hipDeviceSynchronize();

		err = hipGetLastError();
		if (err != hipSuccess)
		{
			printf("Failed to launch kernelVectorAdd (error code %s)!\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}

		err = hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToHost);
		if (err != hipSuccess)
		{
			printf("Failed to copy vector A from device to host (error code %s)!\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}
		err = hipMemcpy(h_B, d_B, size, hipMemcpyDeviceToHost);
		if (err != hipSuccess)
		{
			printf("Failed to copy vector B from device to host (error code %s)!\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}
		err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
		if (err != hipSuccess)
		{
			printf("Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}

		// Verify that the result vector is correct
		printf("\nValidating results ...");
#pragma omp parallel for
		for (int i = 0; i < numElements; ++i)
		{
			if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5)
			{
				printf("Result verification failed at element %d!\n", i);
				exit(EXIT_FAILURE);
			}
		}
		printf(" success!\n");

		err = hipFree(d_A);
		if (err != hipSuccess)
		{
			printf("Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}

		err = hipFree(d_B);
		if (err != hipSuccess)
		{
			printf("Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}

		err = hipFree(d_C);
		if (err != hipSuccess)
		{
			printf("Failed to free device vector C (error code %s)!\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}

		delete[]h_A;
		delete[]h_B;
		delete[]h_C;
	}
	else
	{
		printf("\nExpecting -numele, but saw %s", argv[nextarg]);
		exit(EXIT_FAILURE);
	}
}