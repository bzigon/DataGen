#include "hip/hip_runtime.h"

#include "kernel.h"
#include <hip/hip_runtime.h>
#include <list>
#include <string>
#include <assert.h>
#include <omp.h>

using namespace::std;


typedef struct KernelVectorAddCBTrigILP2_64Params
{
public:
	dim3 m_bs;
	dim3 m_gs;
	int m_NumberOfElements;

	KernelVectorAddCBTrigILP2_64Params(int bsx, int bsy, int bsz, int gsx, int gsy, int gsz, int numele) :
		m_bs(bsx, bsy, bsz),
		m_gs(gsx, gsy, gsz),
		m_NumberOfElements(numele)
	{
		if (bsx < 1) { printf("\n***Error bsx < 1\n"); exit(EXIT_FAILURE); }
		if (bsx > 1024) { printf("\n***Error bsx > 1024\n"); exit(EXIT_FAILURE); }
		if (bsy != 1) { printf("\n***Error bsy != 1\n"); exit(EXIT_FAILURE); }
		if (bsz != 1) { printf("\n***Error bsz != 1\n"); exit(EXIT_FAILURE); }

		if (gsx < 1) { printf("\n***Error gsx < 1\n"); exit(EXIT_FAILURE); }
		if (gsy != 1) { printf("\n***Error gsy != 1\n"); exit(EXIT_FAILURE); }
		if (gsz != 1) { printf("\n***Error gsz != 1\n"); exit(EXIT_FAILURE); }
		if (numele < 1) { printf("\n***Error numele < 1\n"); exit(EXIT_FAILURE); }
	}

} KernelVectorAddCBTrigILP2_64Params_t;

void QueryKernelVectorAddCBTrigILP2_64(char *KernelName, int gs_start, int gs_end, int gs_inc, int numele)
{
	list<KernelVectorAddCBTrigILP2_64Params_t*> params;

	for (int gsx = gs_start; gsx < gs_end; gsx += gs_inc)
		for (int bsx = 32; bsx < 1025; bsx += 32)
			params.push_back(new KernelVectorAddCBTrigILP2_64Params_t(bsx, 1, 1, gsx, 1, 1, numele));

	printf("#\n# %s\n#", KernelName);
	list<KernelVectorAddCBTrigILP2_64Params_t*>::iterator i = params.begin();
	printf("\n%s:   compile: params -bs %4d,%d,%d -gs %4d,%d,%d -numele %d",
		KernelName,
		(*i)->m_bs.x,
		(*i)->m_bs.y,
		(*i)->m_bs.z,
		(*i)->m_gs.x,
		(*i)->m_gs.y,
		(*i)->m_gs.z,
		(*i)->m_NumberOfElements);

	for (i++; i != params.end(); ++i)
	{
		printf("\n%s: nocompile: params -bs %4d,%d,%d -gs %4d,%d,%d -numele %d",
			KernelName,
			(*i)->m_bs.x,
			(*i)->m_bs.y,
			(*i)->m_bs.z,
			(*i)->m_gs.x,
			(*i)->m_gs.y,
			(*i)->m_gs.z,
			(*i)->m_NumberOfElements);
	}
	printf("\n");
}

//
// compute bound version of vector add kernel
//
__global__ void
kernelVectorAddCBTrigILP2_64(float *A, float *B, float *C, float K1, float K2, int numElements)
{
//	const int s1 = blockDim.x * gridDim.x;
	const int stride = blockDim.x * gridDim.x;

	for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < numElements/2; i += stride)
	{
		// C[i] = sin(K1)*A[i] + cos(K2)*B[i]
		// C[i+s] = sin(K1)*A[i+s] + cos(K2)*B[i+s]

		float2 Ar = reinterpret_cast<float2*>(A)[i];	// Ar.x = A[i], Ar.y = A[i+1]
		float2 Br = reinterpret_cast<float2*>(B)[i];	// Br.x = B[i], Br.y = B[i+1]

		float T5 = sin(K1);
		float T6 = cos(K2);
		float T7 = sin(K1);
		float T8 = cos(K2);

		float2 C0, C1;
		C0.x= Ar.x*T5;		// C0.x = A[i] * sin(K1)
		C1.x= Br.x*T6;		// C1.x = B[i] * cos(K2)

		C0.y = Ar.y*T7;		// C0.y = A[i+1] * sin(K1)
		C1.y = Br.y*T8;		// C1.y = B[i+1] * cos(K2)

		float2 C2;
		C2.x = C0.x + C1.x;
		C2.y = C0.y + C1.y;

		reinterpret_cast<float2*>(C)[i] = C2;
	}

	// Process remaining elements
	for (int i = blockDim.x * blockIdx.x + threadIdx.x + (numElements / 2) * 2; i < numElements; i += blockDim.x * gridDim.x+threadIdx.x)
	{
		// C[i] = sin(K1)*A[i] + cos(K2)*B[i]
		float T1 = A[i];
		float T2 = B[i];

		float T3 = sin(K1);
		float T4 = cos(K2);

		float T5 = T1*T3;
		float T6 = T2*T4;

		C[i] = T5 + T6;
	}
}

void LaunchKernelVectorAddCBTrigILP2_64(dim3& gs, dim3& bs, char **argv, int argc, int nextarg)
{
	printf("\nPreparing %s", KernelVectorAddCBTrigILP2_64Name);
	if (strcmp(argv[nextarg], "-numele") == 0)
	{
		printf("\nAllocating RAM");

		hipError_t err = hipSuccess;
		int numElements = stoi(argv[nextarg + 1], nullptr);
		size_t size = numElements * sizeof(float);

		KernelVectorAddCBTrigILP2_64Params_t Verify(bs.x, bs.y, bs.z, gs.x, gs.y, gs.z, numElements);

		float *h_B = new float[numElements];
		float *h_C = new float[numElements];

		// Verify that allocations succeeded
		if (h_B == NULL || h_C == NULL)
		{
			printf("Failed to allocate host vectors in LaunchKernelVectorAddCBTrigILP2_64\n");
			exit(EXIT_FAILURE);
		}

		float *d_A = NULL;
		err = hipMalloc((void **)&d_A, size);
		if (err != hipSuccess)
		{
			printf("Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}

		float *d_B = NULL;
		err = hipMalloc((void **)&d_B, size);
		if (err != hipSuccess)
		{
			printf("Failed to allocate device vector B (error code %s)!\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}

		float *d_C = NULL;
		err = hipMalloc((void **)&d_C, size);
		if (err != hipSuccess)
		{
			printf("Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}

		printf("\nInitializing GPU RAM");
		InitRandomSequence(d_A, numElements);
		InitRandomSequence(d_B, numElements);

		printf("\nLaunching kernel: kernelVectorAddCBTrigILP2_64");
		printf("\n\tgridsize  (%d,%d,%d)", gs.x, gs.y, gs.z);
		printf("\n\tblocksize (%d,%d,%d)", bs.x, bs.y, bs.z);
		printf("\n\tNumElements %d", numElements);

		kernelVectorAddCBTrigILP2_64 << <gs, bs >> > (d_A, d_B, d_C, 0.0f, 0.0f, numElements);
		hipDeviceSynchronize();

		err = hipGetLastError();
		if (err != hipSuccess)
		{
			printf("Failed to launch kernelVectorAddCBTrigILP2_64 (error code %s)!\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}

		err = hipMemcpy(h_B, d_B, size, hipMemcpyDeviceToHost);
		if (err != hipSuccess)
		{
			printf("Failed to copy vector B from device to host (error code %s)!\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}
		err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
		if (err != hipSuccess)
		{
			printf("Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}

		// Verify that the result vector is correct
		printf("\nValidating results ...");
#pragma omp parallel for
		for (int i = 0; i < numElements; ++i)
		{
			if (fabs(h_B[i] - h_C[i]) > 1e-5)
			{
				printf("Result verification failed at element %d!\n", i);
				exit(EXIT_FAILURE);
			}
		}
		printf(" success!\n");

		err = hipFree(d_A);
		if (err != hipSuccess)
		{
			printf("Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}

		err = hipFree(d_B);
		if (err != hipSuccess)
		{
			printf("Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}

		err = hipFree(d_C);
		if (err != hipSuccess)
		{
			printf("Failed to free device vector C (error code %s)!\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}

		delete[]h_B;
		delete[]h_C;
	}
	else
	{
		printf("\nExpecting -numele, but saw %s", argv[nextarg]);
		exit(EXIT_FAILURE);
	}
}