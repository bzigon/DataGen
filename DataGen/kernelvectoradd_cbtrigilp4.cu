#include "hip/hip_runtime.h"

#include "kernel.h"
#include <hip/hip_runtime.h>
#include <list>
#include <string>
#include <assert.h>
#include <omp.h>

using namespace::std;


typedef struct KernelVectorAddCBTrigILP4Params
{
public:
	dim3 m_bs;
	dim3 m_gs;
	int m_NumberOfElements;

	KernelVectorAddCBTrigILP4Params(int bsx, int bsy, int bsz, int gsx, int gsy, int gsz, int numele) :
		m_bs(bsx, bsy, bsz),
		m_gs(gsx, gsy, gsz),
		m_NumberOfElements(numele)
	{
		if (bsx < 1) { printf("\n***Error bsx < 1\n"); exit(EXIT_FAILURE); }
		if (bsx > 1024) { printf("\n***Error bsx > 128\n"); exit(EXIT_FAILURE); }
		if (bsy != 1) { printf("\n***Error bsy != 1\n"); exit(EXIT_FAILURE); }
		if (bsz != 1) { printf("\n***Error bsz != 1\n"); exit(EXIT_FAILURE); }

		if (gsx < 1) { printf("\n***Error gsx < 1\n"); exit(EXIT_FAILURE); }
		if (gsy != 1) { printf("\n***Error gsy != 1\n"); exit(EXIT_FAILURE); }
		if (gsz != 1) { printf("\n***Error gsz != 1\n"); exit(EXIT_FAILURE); }
		if (numele < 1) { printf("\n***Error numele < 1\n"); exit(EXIT_FAILURE); }
	}

} KernelVectorAddCBTrigILP4Params_t;

void QueryKernelVectorAddCBTrigILP4(char *KernelName, int gs_start, int gs_end, int gs_inc, int numele)
{
	list<KernelVectorAddCBTrigILP4Params_t*> params;

	for (int gsx = gs_start; gsx < gs_end; gsx += gs_inc)
		for (int bsx = 32; bsx < 129; bsx += 32)
			params.push_back(new KernelVectorAddCBTrigILP4Params_t(bsx, 1, 1, gsx, 1, 1, numele));

	printf("#\n# %s\n#", KernelName);
	list<KernelVectorAddCBTrigILP4Params_t*>::iterator i = params.begin();
	printf("\n%s:   compile: params -bs %4d,%d,%d -gs %4d,%d,%d -numele %d",
		KernelName,
		(*i)->m_bs.x,
		(*i)->m_bs.y,
		(*i)->m_bs.z,
		(*i)->m_gs.x,
		(*i)->m_gs.y,
		(*i)->m_gs.z,
		(*i)->m_NumberOfElements);

	for (i++; i != params.end(); ++i)
	{
		printf("\n%s: nocompile: params -bs %4d,%d,%d -gs %4d,%d,%d -numele %d",
			KernelName,
			(*i)->m_bs.x,
			(*i)->m_bs.y,
			(*i)->m_bs.z,
			(*i)->m_gs.x,
			(*i)->m_gs.y,
			(*i)->m_gs.z,
			(*i)->m_NumberOfElements);
	}
	printf("\n");
}

//
// compute bound version of vector add kernel
//
#if 0
//
// This version causes the compiler to insert SYNC calls into the code.
__global__ void
kernelVectorAddCBTrigILP4(const float *A, const float *B, float *C, float K1, float K2, int numElements)
{
	const int s1 = blockDim.x * gridDim.x;
	const int s2 = blockDim.x * gridDim.x * 2;
	const int s3 = blockDim.x * gridDim.x * 3;
	const int stride = blockDim.x * gridDim.x * 4;

	for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < numElements; i += stride)
	{
		// C[i]    = sin(K1)*A[i]    + cos(K2)*B[i]
		// C[i+s]  = sin(K1)*A[i+s]  + cos(K2)*B[i+s]
		// C[i+2s] = sin(K1)*A[i+2s] + cos(K2)*B[i+2s]
		// C[i+3s] = sin(K1)*A[i+3s] + cos(K2)*B[i+3s]
		float T1 = A[i];
		float T2 = B[i];
		float T3 = i + s1 < numElements ? A[i + s1] : 0;
		float T4 = i + s1 < numElements ? B[i + s1] : 0;
		float T5 = i + s2 < numElements ? A[i + s2] : 0;
		float T6 = i + s2 < numElements ? B[i + s2] : 0;
		float T7 = i + s3 < numElements ? A[i + s3] : 0;
		float T8 = i + s3 < numElements ? B[i + s3] : 0;

		float T9  = sin(K1);
		float T10 = cos(K2);
		float T11 = sin(K1);
		float T12 = cos(K2);
		float T13 = sin(K1);
		float T14 = cos(K2);
		float T15 = sin(K1);
		float T16 = cos(K2);

		float T17 = T1*T9;
		float T18 = T2*T10;
		float T19 = T3*T11;
		float T20 = T4*T12;
		float T21 = T5*T13;
		float T22 = T6*T14;
		float T23 = T7*T15;
		float T24 = T8*T16;

		C[i]       = T17 + T18;
		if (i + s1 < numElements) C[i + s1]  = T19 + T20;
		if (i + s2 < numElements) C[i + s2] = T21 + T22;
		if (i + s3 < numElements) C[i + s3] = T23 + T24;
	}
}
#endif

#if 1
//

__global__ void
kernelVectorAddCBTrigILP4(const float *A, const float *B, float *C, float K1, float K2, int numElements)
{
	const int s1 = blockDim.x * gridDim.x;
	const int s2 = blockDim.x * gridDim.x * 2;
	const int s3 = blockDim.x * gridDim.x * 3;
	const int stride = blockDim.x * gridDim.x * 4;

	for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < numElements; i += stride)
	{
		// C[i]    = sin(K1)*A[i]    + cos(K2)*B[i]
		// C[i+s]  = sin(K1)*A[i+s]  + cos(K2)*B[i+s]
		// C[i+2s] = sin(K1)*A[i+2s] + cos(K2)*B[i+2s]
		// C[i+3s] = sin(K1)*A[i+3s] + cos(K2)*B[i+3s]
		float T1 = A[i];
		float T2 = B[i];
		float T3 = 0;
		if (i + s1 < numElements)
			T3 = A[i + s1];

		float T4 = 0;
		if (i + s1 < numElements)
			T4 = B[i + s1];

		float T5 = 0;
		if (i + s2 < numElements)
			T5 = A[i + s2];

		float T6 = 0;
		if (i + s2 < numElements)
			T6 = B[i + s2];

		float T7 = 0;
		if (i + s3 < numElements)
			T7 = A[i + s3];

		float T8 = 0;
		if (i + s3 < numElements)
			T8 = B[i + s3];
//		__syncthreads();


		float T9 = sin(K1);
		float T10 = cos(K2);
		float T11 = sin(K1);
		float T12 = cos(K2);
		float T13 = sin(K1);
		float T14 = cos(K2);
		float T15 = sin(K1);
		float T16 = cos(K2);

		float T17 = T1*T9;
		float T18 = T2*T10;
		float T19 = T3*T11;
		float T20 = T4*T12;
		float T21 = T5*T13;
		float T22 = T6*T14;
		float T23 = T7*T15;
		float T24 = T8*T16;

		C[i] = T17 + T18;
		if (i + s1 < numElements) C[i + s1] = T19 + T20;
		if (i + s2 < numElements) C[i + s2] = T21 + T22;
		if (i + s3 < numElements) C[i + s3] = T23 + T24;
	}
}
#endif

#if 0
__global__ void
kernelVectorAddCBTrigILP4(const float *A, const float *B, float *C, float K1, float K2, int numElements)
{
	const int s1 = blockDim.x * gridDim.x;
	const int s2 = blockDim.x * gridDim.x * 2;
	const int s3 = blockDim.x * gridDim.x * 3;
	const int stride = blockDim.x * gridDim.x * 4;

	for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < numElements; i += stride)
	{
		C[i] = sin(K1)*A[i] + cos(K2)*B[i];
		if (i + s1 < numElements) C[i + s1] = sin(K1)*A[i + s1] + cos(K2)*B[i + s1];
		if (i + s2 < numElements) C[i + s2] = sin(K1)*A[i + s2] + cos(K2)*B[i + s2];
		if (i + s3 < numElements) C[i + s3] = sin(K1)*A[i + s3] + cos(K2)*B[i + s3];
	}
}
#endif

#if 0
__global__ void
kernelVectorAddCBTrigILP4(float *A, float *B, float *C, float K1, float K2, int numElements)
{
	const int stride = blockDim.x * gridDim.x;

	for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < numElements/4; i += stride)
	{
		float4 Ar = reinterpret_cast<float4*>(A)[i];	// Ar.x = A[i], Ar.y = A[i+1], Ar.z = A[i+2], Ar.w = A[i+3]
		float4 Br = reinterpret_cast<float4*>(B)[i];	// Br.x = B[i], Br.y = B[i+1], Br.z = B[i+2], Br.w = B[i+3]

		float T5 = sin(K1);
		float T6 = cos(K2);
		float T7 = sin(K1);
		float T8 = cos(K2);
		float T9 = sin(K1);
		float T10 = cos(K2);
		float T11 = sin(K1);
		float T12 = cos(K2);

		float4 C0, C1;
		C0.x = Ar.x*T5;
		C1.x = Br.x*T6;

		C0.y = Ar.y*T7;
		C1.y = Br.y*T8;

		C0.z = Ar.z*T9;
		C1.z = Br.z*T10;

		C0.w = Ar.w*T11;
		C1.w = Br.w*T12;

		float4 C2;
		C2.x = C0.x + C1.x;
		C2.y = C0.y + C1.y;
		C2.z = C0.z + C1.z;
		C2.w = C0.w + C1.w;

		reinterpret_cast<float4*>(C)[i] = C2;
	}

	// Process remaining elements
	for (int i = blockDim.x * blockIdx.x + threadIdx.x + (numElements / 4) * 4; i < numElements; i += blockDim.x * gridDim.x + threadIdx.x)
	{
		C[i] = sin(K1)*A[i] + cos(K2)*B[i];
	}
}
#endif

void LaunchKernelVectorAddCBTrigILP4(dim3& gs, dim3& bs, char **argv, int argc, int nextarg)
{
	printf("\nPreparing %s", KernelVectorAddCBTrigILP4Name);
	if (strcmp(argv[nextarg], "-numele") == 0)
	{
		printf("\nAllocating RAM");

		hipError_t err = hipSuccess;
		int numElements = stoi(argv[nextarg + 1], nullptr);
		size_t size = numElements * sizeof(float);

		KernelVectorAddCBTrigILP4Params_t Verify(bs.x, bs.y, bs.z, gs.x, gs.y, gs.z, numElements);

		float *h_B = new float[numElements];
		float *h_C = new float[numElements];

		// Verify that allocations succeeded
		if (h_B == NULL || h_C == NULL)
		{
			printf("Failed to allocate host vectors in LaunchKernelVectorAddCBTrigILP4\n");
			exit(EXIT_FAILURE);
		}

		float *d_A = NULL;
		err = hipMalloc((void **)&d_A, size);
		if (err != hipSuccess)
		{
			printf("Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}

		float *d_B = NULL;
		err = hipMalloc((void **)&d_B, size);
		if (err != hipSuccess)
		{
			printf("Failed to allocate device vector B (error code %s)!\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}

		float *d_C = NULL;
		err = hipMalloc((void **)&d_C, size);
		if (err != hipSuccess)
		{
			printf("Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}

		printf("\nInitializing GPU RAM");
		InitRandomSequence(d_A, numElements);
		InitRandomSequence(d_B, numElements);

		printf("\nLaunching kernel: kernelVectorAddCBTrigILP4");
		printf("\n\tgridsize  (%d,%d,%d)", gs.x, gs.y, gs.z);
		printf("\n\tblocksize (%d,%d,%d)", bs.x, bs.y, bs.z);
		printf("\n\tNumElements %d", numElements);

		kernelVectorAddCBTrigILP4 << <gs, bs >> > (d_A, d_B, d_C, 0.0f, 0.0f, numElements);
		hipDeviceSynchronize();

		err = hipGetLastError();
		if (err != hipSuccess)
		{
			printf("Failed to launch kernelVectorAddCBTrigILP4 (error code %s)!\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}

		err = hipMemcpy(h_B, d_B, size, hipMemcpyDeviceToHost);
		if (err != hipSuccess)
		{
			printf("Failed to copy vector B from device to host (error code %s)!\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}

		err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
		if (err != hipSuccess)
		{
			printf("Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}

		// Verify that the result vector is correct
		printf("\nValidating results ...");
#pragma omp parallel for
		for (int i = 0; i < numElements; ++i)
		{
			if (fabs(h_B[i] - h_C[i]) > 1e-5)
			{
				printf("Result verification failed at element %d!\n", i);
				exit(EXIT_FAILURE);
			}
		}
		printf(" success!\n");

		err = hipFree(d_A);
		if (err != hipSuccess)
		{
			printf("Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}

		err = hipFree(d_B);
		if (err != hipSuccess)
		{
			printf("Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}

		err = hipFree(d_C);
		if (err != hipSuccess)
		{
			printf("Failed to free device vector C (error code %s)!\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}

		delete[]h_B;
		delete[]h_C;
	}
	else
	{
		printf("\nExpecting -numele, but saw %s", argv[nextarg]);
		exit(EXIT_FAILURE);
	}
}