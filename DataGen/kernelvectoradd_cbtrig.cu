#include "hip/hip_runtime.h"

#include "kernel.h"
#include <hip/hip_runtime.h>
#include <list>
#include <string>
#include <assert.h>
#include <omp.h>

using namespace::std;


typedef struct KernelVectorAddCBTrigParams
{
public:
	dim3 m_bs;
	dim3 m_gs;
	int m_NumberOfElements;

	KernelVectorAddCBTrigParams(int bsx, int bsy, int bsz, int gsx, int gsy, int gsz, int numele) :
		m_bs(bsx, bsy, bsz),
		m_gs(gsx, gsy, gsz),
		m_NumberOfElements(numele)
	{
		if (bsx < 1) { printf("\n***Error bsx < 1\n"); exit(EXIT_FAILURE); }
		if (bsx > 1024) { printf("\n***Error bsx > 1024\n"); exit(EXIT_FAILURE); }
		if (bsy != 1) { printf("\n***Error bsy != 1\n"); exit(EXIT_FAILURE); }
		if (bsz != 1) { printf("\n***Error bsz != 1\n"); exit(EXIT_FAILURE); }

		if (gsx < 1) { printf("\n***Error gsx < 1\n"); exit(EXIT_FAILURE); }
		if (gsy != 1) { printf("\n***Error gsy != 1\n"); exit(EXIT_FAILURE); }
		if (gsz != 1) { printf("\n***Error gsz != 1\n"); exit(EXIT_FAILURE); }
		if (numele < 1) { printf("\n***Error numele < 1\n"); exit(EXIT_FAILURE); }
	}

} KernelVectorAddCBTrigParams_t;

void QueryKernelVectorAddCBTrig(char *KernelName, int bs_start, int bs_end, int bs_inc, int gs_start, int gs_end, int gs_inc, int numele)
{

	list<KernelVectorAddCBTrigParams_t*> params;

	for (int gsx = gs_start; gsx < gs_end; gsx += gs_inc)
		for (int bsx = bs_start; bsx < bs_end; bsx += bs_inc)
			params.push_back(new KernelVectorAddCBTrigParams_t(bsx, 1, 1, gsx, 1, 1, numele));


	printf("#\n# %s\n#", KernelName);
	list<KernelVectorAddCBTrigParams_t*>::iterator i = params.begin();
	printf("\n%s:   compile: params -bs %4d,%d,%d -gs %4d,%d,%d -numele %d",
		KernelName,
		(*i)->m_bs.x,
		(*i)->m_bs.y,
		(*i)->m_bs.z,
		(*i)->m_gs.x,
		(*i)->m_gs.y,
		(*i)->m_gs.z,
		(*i)->m_NumberOfElements);

	for (i++; i != params.end(); ++i)
	{
		printf("\n%s: nocompile: params -bs %4d,%d,%d -gs %4d,%d,%d -numele %d",
			KernelName,
			(*i)->m_bs.x,
			(*i)->m_bs.y,
			(*i)->m_bs.z,
			(*i)->m_gs.x,
			(*i)->m_gs.y,
			(*i)->m_gs.z,
			(*i)->m_NumberOfElements);
	}
	printf("\n");
}

//
// compute bound version of vector add kernel
//
__global__ void
kernelVectorAddCBTrig(const float *A, const float *B, float *C, float K1, float K2, int numElements)
{
	const int stride = blockDim.x * gridDim.x;

	for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < numElements; i += stride)
	{
	//	C[i] = sin(K1)*A[i] + cos(K2)*B[i];
		float T1 = A[i];
		float T2 = B[i];

		float T3 = sin(K1);
		float T4 = cos(K2);

		float T5 = T1*T3;
		float T6 = T2*T4;

		C[i] = T5 + T6;
	}
}

void LaunchKernelVectorAddCBTrig(dim3& gs, dim3& bs, char **argv, int argc, int nextarg)
{
	printf("\nPreparing %s", KernelVectorAddCBTrigName);
	if (strcmp(argv[nextarg], "-numele") == 0)
	{
		printf("\nAllocating RAM");

		hipError_t err = hipSuccess;
		int numElements = stoi(argv[nextarg + 1], nullptr);
		size_t size = numElements * sizeof(float);

		KernelVectorAddCBTrigParams_t Verify(bs.x, bs.y, bs.z, gs.x, gs.y, gs.z, numElements);

		float *h_B = new float[numElements];
		float *h_C = new float[numElements];

		// Verify that allocations succeeded
		if (h_B == NULL || h_C == NULL)
		{
			printf("Failed to allocate host vectors in LaunchKernelVectorAddCBTrig\n");
			exit(EXIT_FAILURE);
		}

		float *d_A = NULL;
		err = hipMalloc((void **)&d_A, size);
		if (err != hipSuccess)
		{
			printf("Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}

		float *d_B = NULL;
		err = hipMalloc((void **)&d_B, size);
		if (err != hipSuccess)
		{
			printf("Failed to allocate device vector B (error code %s)!\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}

		float *d_C = NULL;
		err = hipMalloc((void **)&d_C, size);
		if (err != hipSuccess)
		{
			printf("Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}

		printf("\nInitializing GPU RAM");
		InitRandomSequence(d_A, numElements);
		InitRandomSequence(d_B, numElements);

		printf("\nLaunching kernel: kernelVectorAddCBTrig");
		printf("\n\tgridsize  (%d,%d,%d)", gs.x, gs.y, gs.z);
		printf("\n\tblocksize (%d,%d,%d)", bs.x, bs.y, bs.z);
		printf("\n\tNumElements %d", numElements);

		kernelVectorAddCBTrig << <gs, bs >> > (d_A, d_B, d_C, 0.0f, 0.0f, numElements);
		hipDeviceSynchronize();

		err = hipGetLastError();
		if (err != hipSuccess)
		{
			printf("Failed to launch kernelVectorAddCBTrig (error code %s)!\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}

		err = hipMemcpy(h_B, d_B, size, hipMemcpyDeviceToHost);
		if (err != hipSuccess)
		{
			printf("Failed to copy vector B from device to host (error code %s)!\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}
		err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
		if (err != hipSuccess)
		{
			printf("Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}

		// Verify that the result vector is correct
		printf("\nValidating results ...");
#pragma omp parallel for
		for (int i = 0; i < numElements; ++i)
		{
			if (fabs(h_B[i] - h_C[i]) > 1e-5)
			{
				printf("Result verification failed at element %d!\n", i);
				exit(EXIT_FAILURE);
			}
		}
		printf(" success!\n");

		err = hipFree(d_A);
		if (err != hipSuccess)
		{
			printf("Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}

		err = hipFree(d_B);
		if (err != hipSuccess)
		{
			printf("Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}

		err = hipFree(d_C);
		if (err != hipSuccess)
		{
			printf("Failed to free device vector C (error code %s)!\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}

		delete[]h_B;
		delete[]h_C;
	}
	else
	{
		printf("\nExpecting -numele, but saw %s", argv[nextarg]);
		exit(EXIT_FAILURE);
	}
}